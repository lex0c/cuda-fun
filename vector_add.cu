#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel function to add two vectors
__global__ void vector_add(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1 << 20; // Vector size: 1 million elements

    // Allocate memory on the host
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // Initialize input vectors A and B
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    // Allocate memory on the GPU
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Copy input data from the host to the GPU
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    // Define kernel configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Execute the CUDA kernel
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy the output data from the GPU to the host
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // Verify that the vector addition was performed correctly
    for (int i = 0; i < N; i++) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            std::cerr << "Error: vector addition was not performed correctly." << std::endl;
            return 1;
        }
    }

    // Free memory allocated on the host and the GPU
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    std::cout << "Vector addition completed successfully." << std::endl;
    return 0;
}

